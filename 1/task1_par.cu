#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void parallel_sum_kernel(int *array, long long *result, int size) {
    extern __shared__ long long sdata[];
    
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    
    // Локальное суммирование
    long long local_sum = 0;
    while (i < size) {
        local_sum += array[i];
        i += blockDim.x * gridDim.x;
    }
    sdata[tid] = local_sum;
    __syncthreads();
    
    // Сокращение внутри блока
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    // Запись результата блока
    if (tid == 0) {
        result[blockIdx.x] = sdata[0];
    }
}

int main() {
    const char *env_size = getenv("ARRAY_SIZE");
    int size = env_size ? atoi(env_size) : 200000;
    
    // Конфигурация выполнения
    int threads_per_block = 256;
    int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;
    if (blocks_per_grid > 65536) blocks_per_grid = 65536;
    
    int *d_array;
    long long *d_block_sums;
    long long *h_block_sums = (long long*)malloc(blocks_per_grid * sizeof(long long));
    
    // Выделение памяти на устройстве
    hipMalloc((void**)&d_array, size * sizeof(int));
    hipMalloc((void**)&d_block_sums, blocks_per_grid * sizeof(long long));
    
    // Инициализация массива на хосте
    int *h_array = (int*)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        h_array[i] = i + 1;
    }
    
    // Копирование данных на устройство
    hipMemcpy(d_array, h_array, size * sizeof(int), hipMemcpyHostToDevice);
    
    clock_t start = clock();
    
    // Запуск параллельного ядра
    parallel_sum_kernel<<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(long long)>>>(d_array, d_block_sums, size);
    
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    // Копирование частичных сумм
    hipMemcpy(h_block_sums, d_block_sums, blocks_per_grid * sizeof(long long), hipMemcpyDeviceToHost);
    
    // Финальное суммирование на хосте
    long long final_sum = 0;
    for (int i = 0; i < blocks_per_grid; i++) {
        final_sum += h_block_sums[i];
    }
    
    clock_t end = clock();
    double time_taken = (double)(end - start) / CLOCKS_PER_SEC;
    
    printf("Сумма массива: %lld\n", final_sum);
    printf("Время выполнения (CUDA parallel): %f секунд\n", time_taken);
    
    // Освобождение памяти
    hipFree(d_array);
    hipFree(d_block_sums);
    free(h_array);
    free(h_block_sums);
    
    return 0;
}
