#include <iostream>
#include <hip/hip_runtime.h>

__global__ void elementwise_ops(const float* A, const float* B, float* add, float* sub, float* mul, float* div, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        add[idx] = A[idx] + B[idx];
        sub[idx] = A[idx] - B[idx];
        mul[idx] = A[idx] * B[idx];
        div[idx] = A[idx] / B[idx];
    }
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Использование: " << argv[0] << " <array_size> <threads_per_block>\n";
        return 1;
    }

    int n = std::stoi(argv[1]);
    int threadsPerBlock = std::stoi(argv[2]);
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    size_t size = n * sizeof(float);

    float *A, *B, *add, *sub, *mul, *div;
    float *d_A, *d_B, *d_add, *d_sub, *d_mul, *d_div;

    A = new float[n]; B = new float[n];
    add = new float[n]; sub = new float[n]; mul = new float[n]; div = new float[n];

    for (int i = 0; i < n; ++i) {
        A[i] = 1.5f;
        B[i] = 2.5f;
    }

    hipMalloc(&d_A, size); hipMalloc(&d_B, size);
    hipMalloc(&d_add, size); hipMalloc(&d_sub, size);
    hipMalloc(&d_mul, size); hipMalloc(&d_div, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);

    elementwise_ops<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_add, d_sub, d_mul, d_div, n);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, stop, start);

    std::cout << "Время выполнения CUDA: " << ms / 1000.0 << " секунд\n";

    hipMemcpy(add, d_add, size, hipMemcpyDeviceToHost);
    hipMemcpy(sub, d_sub, size, hipMemcpyDeviceToHost);
    hipMemcpy(mul, d_mul, size, hipMemcpyDeviceToHost);
    hipMemcpy(div, d_div, size, hipMemcpyDeviceToHost);

    delete[] A; delete[] B; delete[] add; delete[] sub; delete[] mul; delete[] div;
    hipFree(d_A); hipFree(d_B); hipFree(d_add); hipFree(d_sub); hipFree(d_mul); hipFree(d_div);

    return 0;
}
